#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloWorld() {
    printf("I am block (%d, %d, %d), thread (%d, %d, %d)\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

int main() {
    dim3 grid(3, 3, 3);
    dim3 block(3, 3, 3);

    helloWorld<<<grid, block>>>();

    hipDeviceReset();

    return 0;
}
