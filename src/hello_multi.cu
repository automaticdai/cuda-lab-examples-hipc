#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloWorld() {
    printf("I am block (%d), thread (%d)\n", blockIdx.x, threadIdx.x);
}

int main() {
    int grid_size = 3;
    int block_size = 3;

    helloWorld<<<grid_size, block_size>>>();

    hipDeviceSynchronize();

    return 0;
}
