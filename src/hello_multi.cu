#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloWorld() {
    printf("I am block (%d), thread (%d)\n", blockIdx.x, threadIdx.x);
}

int main() {
    int grid = 3;
    int block = 3;

    helloWorld<<<grid, block>>>();

    hipDeviceReset();

    return 0;
}
