#include <stdio.h>
#include <hip/hip_runtime.h>

void queryDevice() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Number of GPUs: %d\n", deviceCount);

    int drvVer;
    int runVer;
    hipDriverGetVersion(&drvVer);
    hipRuntimeGetVersion(&runVer);
    hipDeviceProp_t deviceProperties;
    printf("CUDA Driver Version / Runtime Version: %d.%d / %d.%d\n", drvVer/1000, (drvVer%100)/10, runVer/1000, (runVer%100)/10);

    int i;
    for (i = 0; i < deviceCount; i++) {
        hipGetDeviceProperties(&deviceProperties, i);
        printf("Name: %s\n", deviceProperties.name);
    }
}

int main() {
    queryDevice();
}
