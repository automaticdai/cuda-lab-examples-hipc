#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        assert(result == cudaSuccess);
    }
#endif
    return result;
}

__global__ void stride(double* a, int s)
{
    int i = (blockDim.x * blockIdx.x + threadIdx.x) * s;
    a[i] = a[i] + 1;
}

void runTest(int deviceId, int nMB)
{
    int blockSize = 256;
    float ms;

    double *d_a;
    hipEvent_t startEvent, stopEvent;
      
    int n = nMB*1024*1024/sizeof(double);

    // NB:  d_a(33*nMB) for stride case
    checkCuda( hipMalloc(&d_a, n * 33 * sizeof(double)) );

    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );

    printf("Stride, Bandwidth (GB/s):\n");

    stride<<<n/blockSize, blockSize>>>(d_a, 1); // warm up
    
    for (int i = 1; i <= 32; i++) {
        checkCuda( hipMemset(d_a, 0, n * sizeof(double)) );

        checkCuda( hipEventRecord(startEvent,0) );
        stride<<<n/blockSize, blockSize>>>(d_a, i);
        checkCuda( hipEventRecord(stopEvent,0) );
        checkCuda( hipEventSynchronize(stopEvent) );

        checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
        printf("%d\t%f\n", i, 2*nMB/ms);
    }

    checkCuda( hipEventDestroy(startEvent) );
    checkCuda( hipEventDestroy(stopEvent) );
    hipFree(d_a);
}

int main(int argc, char **argv)
{
    int nMB = 4;
    int deviceId = 0;

    hipDeviceProp_t prop;

    checkCuda( hipSetDevice(deviceId) );
    checkCuda( hipGetDeviceProperties(&prop, deviceId) );
    printf("Device: %s\n", prop.name);
    printf("Transfer size (MB): %d\n", nMB);

    runTest(deviceId, nMB);
}
